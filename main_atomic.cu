#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_cooperative_groups.h>

using namespace cooperative_groups;

__device__ int monitor;
__device__ int signal;

__device__ int threadNum;

__global__ void vecAdd(int m, int n, float* A, float* B, float* C ){
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	int jj = threadIdx.x;
	int ii = threadIdx.y;

	int blockNum = gridDim.x * gridDim.y * gridDim.z;

	if ((i==0)&&(j==0)){
		printf("blockNum = %d\n", blockNum);
	}
	
	atomicAdd(&threadNum, 1);
	printf("jj= %d, ii= %d, threadNum = %d, blockNum = %d\n", jj, ii, threadNum, blockNum);

	int count = 0;

	while (count<1){
		count++;
	//	printf("A[%d][%d]\n",i,j);

		// barrier
		if ((ii==0)&&(jj==0)){
			atomicAdd(&monitor, 1);

			printf("monitor = %d\n", monitor);

			if (atomicCAS(&monitor, blockNum, 0)==blockNum){
				atomicCAS(&signal, 0, 1);
				printf("now signal is %d and monitor is %d\n", signal, monitor);	
			}			
			while(atomicCAS(&signal,0,0)==0);
		}

		__syncthreads();
	
		if ((ii==0)&&(jj==0)){
			atomicAdd(&monitor, 1);
			if (atomicCAS(&monitor, blockNum, 0)==blockNum){
				atomicCAS(&signal, 1, 0);
				printf("@@@ now signal is %d and monitor is %d\n", signal, monitor);
			}
			while(atomicCAS(&signal,1,1)==1);
		}

		__syncthreads();


		if ((i<m)&&(j<n)) {
			C[i*n+j] = A[i*n+j]+B[i*n+j];
	//		printf("A[%d][%d]=%f\n",i,j,A[i*n+j]);
		}
		

	}
	

}

int main(){
	int m = 32;
	int n = 32;
	float* h_a = NULL;
	float* h_b = NULL;
	float* h_c = NULL;
	float* d_a = NULL;
	float* d_b = NULL;
	float* d_c = NULL;
	h_a = (float*)malloc(m*n*sizeof(float));
	h_b = (float*)malloc(m*n*sizeof(float));
	h_c = (float*)malloc(m*n*sizeof(float));

	hipMalloc((void**)&d_a, m*n*sizeof(float));
	hipMalloc((void**)&d_b, m*n*sizeof(float));
	hipMalloc((void**)&d_c, m*n*sizeof(float));

	if ((h_a==NULL)||(h_b==NULL)||(h_c==NULL)||(d_a==NULL)||(d_b==NULL)||(d_c==NULL)){
		printf("cannot allocate memory.\n");
	}
	
	//memset(h_c,0,m*n*sizeof(float));
	for (int i=0; i<m; i++){
		for (int j=0; j<n; j++){
			h_a[i*n+j]=i+j;
			h_b[i*n+j]=i+j;
			h_c[i*n+j]=0;
			//printf("%f,%f,%f\n",h_a[i*n+j],h_b[i*n+j],h_c[i*n+j]);
		}
	}
	
	hipMemcpy(d_a, h_a, m*n*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, m*n*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_c, h_c, m*n*sizeof(float), hipMemcpyHostToDevice);
	dim3 grid((n+15)/16, (m+15)/16,1);
	dim3 block(16, 16,1);
	//vecAdd<<<grid, block>>>	(m, n, d_a, d_b, d_c);
	vecAdd<<<8, 4>>> (m, n, d_a, d_b, d_c);
	hipMemcpy(h_c, d_c, m*n*sizeof(float), hipMemcpyDeviceToHost);
	for (int i=0; i<m; i++){
		for (int j=0; j<n; j++){
			//printf("C[%d][%d]=%f\n",i,j,h_c[i*n+j]);
		}
	}	
	return 0;	
	
}
