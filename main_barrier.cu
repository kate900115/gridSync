#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_cooperative_groups.h>
#include <chrono>
#include <ctime>
#include <iostream>

using namespace cooperative_groups;

__global__ void vecAdd(int m, int n, float* A, float* B, float* C ){
//	int j = blockIdx.x * blockDim.x + threadIdx.x;
//	int i = blockIdx.y * blockDim.y + threadIdx.y;


	int count = 0;
	while(count<1000000){
		count++;

		//CUDA kernel code here.

		grid_group g = this_grid();
		g.sync();
	}
}

int main(){
	int m = 256;
	int n = 256;
	float* h_a = NULL;
	float* h_b = NULL;
	float* h_c = NULL;
	float* d_a = NULL;
	float* d_b = NULL;
	float* d_c = NULL;
	h_a = (float*)malloc(m*n*sizeof(float));
	h_b = (float*)malloc(m*n*sizeof(float));
	h_c = (float*)malloc(m*n*sizeof(float));

	hipMalloc((void**)&d_a, m*n*sizeof(float));
	hipMalloc((void**)&d_b, m*n*sizeof(float));
	hipMalloc((void**)&d_c, m*n*sizeof(float));

	if ((h_a==NULL)||(h_b==NULL)||(h_c==NULL)||(d_a==NULL)||(d_b==NULL)||(d_c==NULL)){
		printf("cannot allocate memory.\n");
	}
	
	//memset(h_c,0,m*n*sizeof(float));
	for (int i=0; i<m; i++){
		for (int j=0; j<n; j++){
			h_a[i*n+j]=i+j;
			h_b[i*n+j]=i+j;
			h_c[i*n+j]=0;
//			printf("%f,%f,%f\n",h_a[i*n+j],h_b[i*n+j],h_c[i*n+j]);
		}
	}
	
	hipMemcpy(d_a, h_a, m*n*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, m*n*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_c, h_c, m*n*sizeof(float), hipMemcpyHostToDevice);
	dim3 grid((n+15)/16, (m+15)/16,1);
	dim3 block(16, 16,1);

	auto start = std::chrono::high_resolution_clock::now();

	vecAdd<<<grid, block>>>	(m, n, d_a, d_b, d_c);
	hipMemcpy(h_c, d_c, m*n*sizeof(float), hipMemcpyDeviceToHost);

	auto end = std::chrono::high_resolution_clock::now();

	std::chrono::duration<double> diff = end-start;
	std::cout<<"it took me "<<diff.count()<<" seconds."<<std::endl;	

	for (int i=0; i<m; i++){
		for (int j=0; j<n; j++){
//			printf("C[%d][%d]=%f\n",i,j,h_c[i*n+j]);
		}
	}	
	return 0;	
	
}
